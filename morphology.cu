#include "hip/hip_runtime.h"
#include "morphology.h"
#include <hip/hip_runtime.h>


enum class MorphOpType {
    ERODE,
    DILATE,
};


template <MorphOpType opType>
inline __device__ unsigned char elementOp(unsigned char lhs, unsigned char rhs)
{
}

template <>
inline __device__ unsigned char elementOp<MorphOpType::ERODE>(unsigned char lhs, unsigned char rhs)
{
    return min(lhs, rhs);
}

template <>
inline __device__ unsigned char elementOp<MorphOpType::DILATE>(unsigned char lhs, unsigned char rhs)
{
    return max(lhs, rhs);
}


template <MorphOpType opType>
inline __device__ unsigned char borderValue()
{
}

template <>
inline __device__ unsigned char borderValue<MorphOpType::ERODE>()
{
    return 0;
}

template <>
inline __device__ unsigned char borderValue<MorphOpType::DILATE>()
{
    return 255;
}


// NOTE: step-efficient parallel scan
template <MorphOpType opType>
__device__ void reversedScan(unsigned char* __restrict__ buffer,
        unsigned char* __restrict__ opArray,
        const int selSize,
        const int tid)
{
    opArray[tid] = buffer[tid];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2) {
        if (tid <= selSize - 1 - offset) {
            opArray[tid] = elementOp<opType>(opArray[tid], opArray[tid + offset]);
        }
        __syncthreads();
    }
}

// NOTE: step-efficient parallel scan
template <MorphOpType opType>
__device__ void scan(unsigned char* __restrict__ buffer,
        unsigned char* __restrict__ opArray,
        const int selSize,
        const int tid)
{
    opArray[tid] = buffer[tid];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2) {
        if (tid >= offset) {
            opArray[tid] = elementOp<opType>(opArray[tid], opArray[tid - offset]);
        }
        __syncthreads();
    }
}

// NOTE: step-efficient parallel scan
template <MorphOpType opType>
__device__ void twoWayScan(unsigned char* __restrict__ buffer,
        unsigned char* __restrict__ opArray,
        const int selSize,
        const int tid)
{
    opArray[tid] = buffer[tid];
    opArray[tid + selSize] = buffer[tid + selSize];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2) {
        if (tid >= offset) {
            opArray[tid + selSize - 1] = elementOp<opType>(opArray[tid + selSize - 1], opArray[tid + selSize - 1 - offset]);
        }
        if (tid <= selSize - 1 - offset) {
            opArray[tid] = elementOp<opType>(opArray[tid], opArray[tid + offset]);
        }
        __syncthreads();
    }
}


template <MorphOpType opType>
__global__ void vhgw_horiz(unsigned char* __restrict__ dst,
        unsigned char* __restrict__ src,
        const int width,
        const int height,
        const int selSize
        )
{
    extern __shared__ unsigned char sMem[];
    unsigned char* buffer = sMem;
    unsigned char* opArray = buffer + 2 * selSize;

    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx >= width || tidy >= height) {
        return;
    }

    buffer[threadIdx.x] = src[tidy * width + tidx];
    if (tidx + selSize < width) {
        buffer[threadIdx.x + selSize] = src[tidy * width + tidx + selSize];
    }
    __syncthreads();

    // scan<opType>(buffer + selSize - 1, opArray + selSize - 1, selSize, threadIdx.x);
    // reversedScan<opType>(buffer, opArray, selSize, threadIdx.x);
    twoWayScan<opType>(buffer, opArray, selSize, threadIdx.x);

    if (tidx + selSize/2 < width - selSize/2) {
        dst[tidy * width + tidx + selSize/2] = elementOp<opType>(opArray[threadIdx.x], opArray[threadIdx.x + selSize - 1]);
    }
}

template <MorphOpType opType>
__global__ void vhgw_vert(unsigned char* __restrict__ dst,
        unsigned char* __restrict__ src,
        const int width,
        const int height,
        const int selSize
        )
{
    extern __shared__ unsigned char sMem[];
    unsigned char* buffer = sMem;
    unsigned char* opArray = buffer + 2 * selSize;

    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height || tidx >= width) {
        return;
    }

    buffer[threadIdx.y] = src[tidy * width + tidx];
    if (tidy + selSize < height) {
        buffer[threadIdx.y + selSize] = src[(tidy + selSize) * width + tidx];
    }
    __syncthreads();

    // scan<opType>(buffer + selSize - 1, opArray + selSize - 1, selSize, threadIdx.y);
    // reversedScan<opType>(buffer, opArray, selSize, threadIdx.y);
    twoWayScan<opType>(buffer, opArray, selSize, threadIdx.y);

    if (tidy + selSize/2 < height - selSize/2) {
        dst[(tidy + selSize/2) * width + tidx] = elementOp<opType>(opArray[threadIdx.y], opArray[threadIdx.y + selSize - 1]);
    }

    if (tidy < selSize/2 || tidy >= height - selSize/2) {
        dst[tidy * width + tidx] = borderValue<opType>();
    }
}


template <MorphOpType opType>
void morphology(unsigned char* img_d,
        const int width,
        const int height,
        const int hsize,
        const int vsize)
{
    size_t imageMemSize = width * height * sizeof(unsigned char);
    unsigned char* tmp_d;
    hipMalloc((void **) &tmp_d, imageMemSize);

    dim3 blockSize;
    blockSize.x = hsize;
    blockSize.y = 1;
    dim3 gridSize;
    gridSize.x = roundUp(width, blockSize.x);
    gridSize.y = roundUp(height, blockSize.y);
    size_t sMemSize = 4 * hsize * sizeof(unsigned char);
    vhgw_horiz<opType><<<gridSize, blockSize, sMemSize>>>(tmp_d, img_d, width, height, hsize);

    // hipMemset(img_d, 0, imageMemSize);

    blockSize.x = 1;
    blockSize.y = vsize;
    gridSize.x = roundUp(width, blockSize.x);
    gridSize.y = roundUp(height, blockSize.y);
    sMemSize = 4 * vsize * sizeof(unsigned char);
    vhgw_vert<opType><<<gridSize, blockSize, sMemSize>>>(img_d, tmp_d, width, height, vsize);

    hipFree(tmp_d);
}


extern "C"
void erode(unsigned char* img_d,
        const int width,
        const int height,
        const int hsize,
        const int vsize)
{
    morphology<MorphOpType::ERODE>(img_d, width, height, hsize, vsize);
}

extern "C"
void dilate(unsigned char* img_d,
        const int width,
        const int height,
        const int hsize,
        const int vsize)
{
    morphology<MorphOpType::DILATE>(img_d, width, height, hsize, vsize);
}
