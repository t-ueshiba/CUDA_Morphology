#include "hip/hip_runtime.h"
#include "morphology.h"
#include <hip/hip_runtime.h>


enum class MorphOpType
{
    ERODE,
    DILATE,
};


template <MorphOpType opType> inline __device__ unsigned char
elementOp(unsigned char lhs, unsigned char rhs);

template <> inline __device__ unsigned char
elementOp<MorphOpType::ERODE>(unsigned char lhs, unsigned char rhs)
{
    return min(lhs, rhs);
}

template <> inline __device__ unsigned char
elementOp<MorphOpType::DILATE>(unsigned char lhs, unsigned char rhs)
{
    return max(lhs, rhs);
}

template <MorphOpType opType> inline __device__ unsigned char
borderValue();

template <> inline __device__ unsigned char
borderValue<MorphOpType::ERODE>()
{
    return 0;
}

template <> inline __device__ unsigned char
borderValue<MorphOpType::DILATE>()
{
    return 255;
}


// NOTE: step-efficient parallel scan
template <MorphOpType opType> __device__ void
reversedScan(unsigned char* __restrict__ buffer,
	     unsigned char* __restrict__ opArray,
	     int selSize, int tid)
{
    opArray[tid] = buffer[tid];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2)
    {
        if (tid <= selSize - 1 - offset)
            opArray[tid] = elementOp<opType>(opArray[tid],
					     opArray[tid + offset]);
        __syncthreads();
    }
}

// NOTE: step-efficient parallel scan
template <MorphOpType opType> __device__ void
scan(unsigned char* __restrict__ buffer,
     unsigned char* __restrict__ opArray,
     int selSize, int tid)
{
    opArray[tid] = buffer[tid];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2)
    {
        if (tid >= offset)
            opArray[tid] = elementOp<opType>(opArray[tid],
					     opArray[tid - offset]);
	__syncthreads();
    }
}

// NOTE: step-efficient parallel scan
template <MorphOpType opType> __device__ void
twoWayScan(unsigned char* __restrict__ buffer,
	   unsigned char* __restrict__ opArray,
	   int selSize, int tid)
{
    opArray[tid]	   = buffer[tid];
    opArray[tid + selSize] = buffer[tid + selSize];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2)
    {
        if (tid >= offset)
            opArray[tid + selSize - 1]
		= elementOp<opType>(opArray[tid + selSize - 1],
				    opArray[tid + selSize - 1 - offset]);
        if (tid <= selSize - 1 - offset)
            opArray[tid] = elementOp<opType>(opArray[tid],
					     opArray[tid + offset]);
        __syncthreads();
    }
}


template <MorphOpType opType> __global__ void
vhgw_horiz(unsigned char* __restrict__ dst, unsigned char* __restrict__ src,
	   int width, int height, int selSize)
{
    extern __shared__ unsigned char	sMem[];
    unsigned char*			buffer  = sMem;
    unsigned char*			opArray = buffer + 2*selSize;

    const int	u = threadIdx.x + blockIdx.x*blockDim.x;
    const int	v = threadIdx.y + blockIdx.y*blockDim.y;
    if (u >= width || v >= height)
        return;

    buffer[threadIdx.x] = src[v*width + u];
    if (u + selSize < width)
        buffer[threadIdx.x + selSize] = src[v*width + u + selSize];
    __syncthreads();

    // scan<opType>(buffer + selSize - 1, opArray + selSize - 1, selSize, threadIdx.x);
    // reversedScan<opType>(buffer, opArray, selSize, threadIdx.x);
    twoWayScan<opType>(buffer, opArray, selSize, threadIdx.x);

    if (u + selSize/2 < width - selSize/2)
        dst[v*width + u + selSize/2]
	    = elementOp<opType>(opArray[threadIdx.x],
				opArray[threadIdx.x + selSize - 1]);
}

template <MorphOpType opType> __global__ void
vhgw_vert(unsigned char* __restrict__ dst, unsigned char* __restrict__ src,
	  int width, int height, int selSize)
{
    extern __shared__ unsigned char	sMem[];
    unsigned char*			buffer  = sMem;
    unsigned char*			opArray = buffer + 2*selSize;

    const int	u = threadIdx.x + blockIdx.x*blockDim.x;
    const int	v = threadIdx.y + blockIdx.y*blockDim.y;
    if (v >= height || u >= width)
        return;

    buffer[threadIdx.y] = src[v*width + u];
    if (v + selSize < height)
        buffer[threadIdx.y + selSize] = src[(v + selSize)*width + u];
    __syncthreads();

    // scan<opType>(buffer + selSize - 1, opArray + selSize - 1, selSize, threadIdx.y);
    // reversedScan<opType>(buffer, opArray, selSize, threadIdx.y);
    twoWayScan<opType>(buffer, opArray, selSize, threadIdx.y);

    if (v + selSize/2 < height - selSize/2)
        dst[(v + selSize/2)*width + u]
	    = elementOp<opType>(opArray[threadIdx.y],
				opArray[threadIdx.y + selSize - 1]);

    if (v < selSize/2 || v >= height - selSize/2)
        dst[v*width + u] = borderValue<opType>();
}


template <MorphOpType opType> void
morphology(unsigned char* img_d, int width, int height, int hsize, int vsize)
{
    const size_t	imageMemSize = width * height * sizeof(unsigned char);
    unsigned char*	tmp_d;
    hipMalloc((void **) &tmp_d, imageMemSize);

    dim3	blockSize(hsize, 1);
    dim3	gridSize(roundUp(width, blockSize.x),
			 roundUp(height, blockSize.y));
    size_t	sMemSize = 4 * hsize * sizeof(unsigned char);
    vhgw_horiz<opType><<<gridSize, blockSize, sMemSize>>>(tmp_d, img_d,
							  width, height,
							  hsize);

    // hipMemset(img_d, 0, imageMemSize);

    blockSize.x = 1;
    blockSize.y = vsize;
    gridSize.x  = roundUp(width, blockSize.x);
    gridSize.y  = roundUp(height, blockSize.y);
    sMemSize	= 4 * vsize * sizeof(unsigned char);
    vhgw_vert<opType><<<gridSize, blockSize, sMemSize>>>(img_d, tmp_d,
							 width, height,
							 vsize);

    hipFree(tmp_d);
}


extern "C" void
erode(unsigned char* img_d, int width, int height, int hsize, int vsize)
{
    morphology<MorphOpType::ERODE>(img_d, width, height, hsize, vsize);
}

extern "C" void
dilate(unsigned char* img_d, int width, int height, int hsize, int vsize)
{
    morphology<MorphOpType::DILATE>(img_d, width, height, hsize, vsize);
}
